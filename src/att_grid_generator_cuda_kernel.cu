#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>
#include <c10/macros/Macros.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n) \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
         i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;

inline int GET_BLOCKS(const int N)
{
    return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

template <typename scalar_t>
__global__ void att_grid_generator_gpu_kernel(const int n, 
                                              scalar_t *mapx,
                                              scalar_t *mapy, 
                                              scalar_t *map_xi,
                                              scalar_t *map_yi, 
                                              scalar_t *index_x, 
                                              scalar_t *index_y, 
                                              const int batch_size,
                                              const int att_size,
                                              const int out_size,
                                              const float threshold,
                                              const int iters)
{
    CUDA_KERNEL_LOOP(index, n) {
        const int b = index; 
        scalar_t *mapx_ptr = mapx + b * att_size * 1;
        scalar_t *mapy_ptr = mapy + b * att_size * 1;
        scalar_t *map_xi_ptr = map_xi + b * att_size * 1;
        scalar_t *map_yi_ptr = map_yi + b * att_size * 1;
        scalar_t *index_x_ptr = index_x + b * out_size * 1;
        scalar_t *index_y_ptr = index_y + b * out_size * 1;
        scalar_t threshold_s = static_cast<scalar_t>(threshold);
        scalar_t threshold_in_use = threshold_s;
        for (int i = 0; i < att_size; i++) {
           mapx_ptr[i] = mapx_ptr[i] * out_size;
           mapy_ptr[i] = mapy_ptr[i] * out_size; 
        }
        for (int j = 0; j < iters; j++) {
            scalar_t map_max_x = 0;
            scalar_t map_max_y = 0;
            for (int k = 0; k < att_size; k++) {
                map_max_x = map_max_x > mapx_ptr[k] ? map_max_x : mapx_ptr[k];
                map_max_y = map_max_y > mapy_ptr[k] ? map_max_y : mapy_ptr[k];
            }
            map_max_x = map_max_x > map_max_y ? map_max_y : map_max_x;
            threshold_in_use = map_max_x;
            if (j == 0)
                threshold_in_use = threshold_s > map_max_x ? map_max_x : threshold_s;
            for (int k = 0; k < att_size; k++) {
                mapx_ptr[k] = mapx_ptr[k] > threshold_in_use ? threshold_in_use : mapx_ptr[k];
                mapy_ptr[k] = mapy_ptr[k] > threshold_in_use ? threshold_in_use : mapy_ptr[k];
            }

            scalar_t sum_x = 0;
            scalar_t sum_y = 0;
            for (int k = 0; k < att_size; k++) {
                sum_x += mapx_ptr[k];
                sum_y += mapy_ptr[k];
            }

            scalar_t delta_x = (out_size - sum_x) / att_size;
            scalar_t delta_y = (out_size - sum_y) / att_size;

            for (int k = 0; k < att_size; k++) {
                mapx_ptr[k] += delta_x;
                mapy_ptr[k] += delta_y;
            }

        }

        for (int i = 0; i < att_size - 1; i++) {
            map_xi_ptr[i + 1] = map_xi_ptr[i] + mapx_ptr[i + 1];
            map_yi_ptr[i + 1] = map_yi_ptr[i] + mapy_ptr[i + 1];
        }

        scalar_t step_x = map_xi_ptr[att_size - 1] / out_size;
        scalar_t step_y = map_yi_ptr[att_size - 1] / out_size;
        int i = 0; 
        int j = 1;
        scalar_t myscale = 2.0 / (att_size - 1);
        
        while (i < out_size) {
            if (map_xi_ptr[j] >= i * step_x) {
                index_y_ptr[i] = (j + (i * step_x - map_xi_ptr[j]) / (map_xi_ptr[j] - map_xi_ptr[j-1])) * myscale - 1.0;
                i++;
            }
            else
                j++;
        }

        i = 0;
        j = 1;

        while (i < out_size) {
            if (map_yi_ptr[j] >= i * step_y) {
                index_x_ptr[i] = (j + (i * step_y - map_yi_ptr[j]) / (map_yi_ptr[j] - map_yi_ptr[j-1])) * myscale - 1.0;
                i++;
            }
            else    
                j++;
        }
    }
}


void attgridgen_gpu(const at::Tensor attx, const at::Tensor atty,
    at::Tensor map_xi, at::Tensor map_yi,
    at::Tensor index_x, at::Tensor index_y,
    const int batch_size, const int att_size, const int out_size, 
    const float threshold, const int iters)
{
    int num_kernels = batch_size;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        attx.type(), "att_grid_generator_gpu", ([&] {
            scalar_t *attx_ = attx.data_ptr<scalar_t>();
            scalar_t *atty_ = atty.data_ptr<scalar_t>();
            scalar_t *map_xi_ = map_xi.data_ptr<scalar_t>();
            scalar_t *map_yi_ = map_yi.data_ptr<scalar_t>();
            scalar_t *index_x_ = index_x.data_ptr<scalar_t>();
            scalar_t *index_y_ = index_y.data_ptr<scalar_t>();

            att_grid_generator_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
                num_kernels, attx_, atty_, map_xi_, map_yi_, index_x_, index_y_, 
                batch_size, att_size, out_size, threshold, iters);

        })
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in att_grid_generator: %s\n", hipGetErrorString(err));
    }

}
